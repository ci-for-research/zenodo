
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_gpu(void) {
    printf("Houston, we have a problem in section [%d,%d] \
        From Apollo 13\n", threadIdx.x,blockIdx.x);
}

int main(void) {
    printf("This is Houston. Say again, please. \
                From Base\n");
    print_gpu<<<2,2>>>();
    hipDeviceSynchronize();
    return 0;
}
